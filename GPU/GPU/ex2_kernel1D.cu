
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <math.h>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

__global__ void add(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] += x[i];
    }
}

int main(int argc, char const *argv[])
{
    const int N = argc >= 2 ? std::stoi(argv[1]) : 1e6;
    std::cout << "N = " << N << std::endl;

    float *x, *y, *dx, *dy;
    x = (float *) malloc(N * sizeof(float));
    y = (float *) malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    hipMalloc(&dx, N * sizeof(float));
    hipMalloc(&dy, N * sizeof(float));
    hipMemcpy(dx, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, N * sizeof(float), hipMemcpyHostToDevice);
    add<<<(N + 255) / 256, 256>>>(N, dx, dy);
    hipMemcpy(y, dy, N * sizeof(float), hipMemcpyDeviceToHost);
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;
    free(x);
    free(y);
    hipFree(dx);
    hipFree(dy);
    return 0;
}
