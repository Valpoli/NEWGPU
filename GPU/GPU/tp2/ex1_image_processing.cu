#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "image.h"

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

template <typename T>
__device__ inline T* get_ptr(T *img, int i, int j, int C, size_t pitch) {
    return img + i * pitch / sizeof(float) + j * C;
}

__global__ void process(int N, int M, int C, int pitch, float* img)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < M && j < N) {
        float* pixel = get_ptr(img,i,j,C,pitch);
        float newColor = 0;
        for (int k=0; k<C; k+=1)
        {
            newColor += pixel[k];
        }
        newColor =  newColor/C;
        for (int k=0; k<C; k+=1)
        {
            pixel[k] = newColor;
        }
    }
}

int main(int argc, char const *argv[])
{
    const std::string filename = argc >= 2 ? argv[1] : "image.jpg";
    std::cout << "filename = " << filename << std::endl;
    int M = 0;
    int N = 0;
    int C = 0;
    float* img = image::load(filename, &N, &M, &C);
    std::cout << "N (columns, width) = " << N << std::endl;
    std::cout << "M (rows, height) = " << M << std::endl;
    std::cout << "C (channels, depth) = " << C << std::endl;

    size_t pitch;

    float* cpy;
    CUDA_CHECK(hipMallocPitch(&cpy, &pitch, N * C * sizeof(float), M));
    CUDA_CHECK(hipMemcpy2D(cpy, pitch, img, N * C * sizeof(float), N * C * sizeof(float), M, hipMemcpyHostToDevice));
    
    // launch kernel
    dim3 block_dim(16, 16);
    dim3 grid_dim((M + block_dim.x - 1) / block_dim.x, (N + block_dim.y - 1) / block_dim.y);
    process<<<grid_dim, block_dim>>>(N,M,C,pitch,cpy);
    
    // copy device memory back to host memory
    CUDA_CHECK(hipMemcpy2D(img, C * N * sizeof(float), cpy, pitch, C * N * sizeof(float), M, hipMemcpyDeviceToHost));
    image::save("result.jpg", N, M, C, img);

    hipFree(cpy);
    free(img);

    return 0;
}
