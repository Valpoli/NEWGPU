
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}

int main()
{
    int device_count;

    CUDA_CHECK(hipGetDeviceCount(&device_count));

    std::cout << "device count = " << device_count << std::endl;

    for(auto i = 0; i < device_count; ++i)
    {
        // ...
    }

    return 0;
}